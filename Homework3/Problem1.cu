#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ float array_mean(float *x, int n) {
  float sum = 0;
  for (int i = 0; i < n; i++) {
    sum += x[i];
  }
  return sum / n;
}

__global__ void block_burst(float *x, int n, int k, long *meanIdx, float * meanData) {
  int start = threadIdx.x * blockDim.x + threadIdx.y;
  int length = threadIdx.x * blockDim.x + threadIdx.y + k;
  extern __shared__ float currentmax[];
  currentmax[threadIdx.x] = array_mean(x + start, length);
  meanIdx[threadIdx.x] = (x + start) << 32 | length;
  meanData[threadIdx.x] = currentmax[threadIdx.x];
}

void maxburst(float *x, int n, int k, int *startend, float *bigmax) {
  hipblasHandle_t handle;
  float *device_x;
  int maxIdx = 0;
  hipMalloc((void **)&device_x, sizeof(float) * n);
  hipMemcpy(device_x, x, sizeof(float) * n, hipMemcpyHostToDevice);
  int *device_result;
  hipMalloc((void **)&device_result, sizeof(int) * 3);

  long* meanIdx;
  hipMalloc((void **)&meanIdx, sizeof(long) * n);// size change

  float *meanData;
  hipMalloc((void **)&meanData, sizeof(float) * n);// size

  dim3 dimGrid(n - k + 1, 1);
  dim3 dimBlock(n, 1, 1);

  block_burst<<<dimGrid, dimBlock, (n - k + 1) * n>>>(device_x, n, k, meanIdx, meanData);

  hipMemcpy()
  // hipMemcpy(startend, device_result, sizeof(int) * 2, hipMemcpyDeviceToHost);
  // hipMemcpy(bigmax, device_result + 2, sizeof(int) * 1, hipMemcpyDeviceToHost);
  hipMemcpy();
  hipMemcpy();

  hipDeviceSynchronize();
  hipblasStatus_t hipblasIsamax(handle, n, meanData, 0, &maxIdx);
  long index = meanIdx[maxIdx];
  startend[1] = index & 0xffffffff;
  startend[0] = index >> 32;
  *bigmax = meanData[maxIdx];
  hipFree(device_x);
  hipFree(device_result);
}

int main(int argc, char const *argv[]) {
  float x[7] = {1, 10, 3, 4, 3, 8, 1};
  int n = 7;
  int k = 3;
  int startend[2] = {0};
  float bigmax = 0;
  maxburst(x, n, k, startend, &bigmax);
  printf("%f (%d %d)\n", bigmax, startend[0], startend[1]);
  return 0;
}
